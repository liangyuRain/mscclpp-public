#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <mscclpp/sm_channel_device.hpp>
#include <mscclpp/proxy_channel_device.hpp>

// BEGIN_DEFINES //

#ifndef PARAMETRIZE
#define KERNEL pipeline_schedule
#define N_PEERS 8
#endif

#define FLUSH_INTERVAL 50

// END_DEFINES //

/// The call is a single node in the tree.
///
/// Syncronization: It is guaranteed that in allreduce and allgather, after the kernel finishes,
/// the memory is safe to be written. However, the kernel immediately starts to write remote
/// memory once launched. There are two cases:
/// Allreduce/ReduceScatter - The kernel writes to remote scratch buffer in the beginning.
/// As long as the remote peer has finished initializing scratch buffer, the write is safe.
/// Allgather - The kernel (proxy) waits for remote signal before writing to remote.
///
/// @param recv_sm_channels SM channels for recv.
/// @param send_sm_channels SM channels for send.
/// @param recv_proxy_channels Proxy channels for recv.
/// @param send_proxy_channels Proxy channels for send.
/// @param recv_scratches Scratch buffers for each recv_sm_channels, len(recv_scratches) == len(recv_sm_channels) + len(recv_proxy_channels)
/// @param nrecv_sm Num of valid recv_sm_channels (nrecv_sm + nrecv_proxy <= 1 if broadcast).
/// @param nsend_sm Num of valid send_sm_channels (nsend_sm + nsend_proxy <= 1 if reduce).
/// @param nrecv_proxy Num of valid recv_proxy_channels (nrecv_sm + nrecv_proxy <= 1 if broadcast).
/// @param nsend_proxy Num of valid send_proxy_channels (nsend_sm + nsend_proxy <= 1 if reduce).
/// @param node_type <0: reduce node; =0: root node; >0: broadcast node.
/// The send channels of broadcast and root nodes write to data buffer.
/// The send channels of reduce node write to scratch buffer.
/// @param scratch_size Max num of elements in scratch buffer for each recv channel (ignore if not reduce).
/// scratch_size must be greater than nelem_per_send
/// @param data Data buffer.
/// @param data_start The data buffer start.
/// @param nelem_per_send Num of elements in each send.
/// @param nelem_total Total num of elements need to be send/recv.
MSCCLPP_DEVICE_INLINE void 
    threadblockCall(mscclpp::SmChannelDeviceHandle* recv_sm_channels, mscclpp::SmChannelDeviceHandle* send_sm_channels,
                    mscclpp::SimpleProxyChannelDeviceHandle* recv_proxy_channels, mscclpp::SimpleProxyChannelDeviceHandle* send_proxy_channels,
                    int** recv_scratches, const int nrecv_sm, const int nsend_sm, const int nrecv_proxy, const int nsend_proxy,
                    const char node_type, const uint64_t scratch_size, int* data,
                    const uint64_t data_start, const uint64_t nelem_per_send, const uint64_t nelem_total) {
  const int tid = threadIdx.x;

  const int nloops = (nelem_total + nelem_per_send - 1) / nelem_per_send; // ceiling division

  if (node_type <= 0) {
    int reduced[N_PEERS] = {};
    __shared__ int ready[N_PEERS];
    __shared__ int pending_sends; // Only reduce node needs to track pending sends.
                                  // Reduce node has at most one send peer.
    int poll_loop_cnt = 0;

    #pragma unroll
    for (int i = tid; i < N_PEERS; i += blockDim.x) ready[i] = 0;
    if (tid == 0) pending_sends = 0;
    __syncthreads();

    const int max_pending_sends = scratch_size / nelem_per_send;

    int rloop = nrecv_sm + nrecv_proxy > 0 ? 0 : nloops; // progress of recv
    int sloop = nsend_sm + nsend_proxy > 0 ? 0 : nloops; // progress of send
    while (rloop < nloops || sloop < nloops) {
      if (rloop < nloops) {
        // assert nrecv_sm + nrecv_proxy > 0
        for (int i = tid; i < nrecv_sm + nrecv_proxy; i += blockDim.x) {
          const int ready_loop = ready[i];
          // if (ready_loop < rloop + 1) ready[i] += (i < nrecv_sm ? recv_sm_channels[i].poll(rloop + 1 - ready_loop) :
          //                                                         recv_proxy_channels[i - nrecv_sm].poll(rloop + 1 - ready_loop));
          if (ready_loop < nloops) ready[i] += (i < nrecv_sm ? recv_sm_channels[i].poll(nloops - ready_loop) :
                                                               recv_proxy_channels[i - nrecv_sm].poll(nloops - ready_loop));
        }
        __syncthreads();

        int count[N_PEERS] = {};
        rloop = nloops;
        for (int i = 0; i < nrecv_sm + nrecv_proxy; ++i) {
          const int ready_loop = ready[i];
          if (reduced[i] < ready_loop){
            do {
              const uint64_t s_start = (reduced[i] % max_pending_sends) * nelem_per_send;
              const uint64_t d_start = data_start + reduced[i] * nelem_per_send;
              const int diff = min(ready_loop - reduced[i], max_pending_sends - reduced[i] % max_pending_sends);

              const uint64_t nElem = min(nelem_per_send * diff, data_start + nelem_total - d_start);
              const uint64_t nElem4 = nElem / 4;
              const uint64_t nLastElem = nElem % 4;

              int4* const data4 = (int4*) &data[d_start];
              int4* const scratch4 = (int4*) &recv_scratches[i][s_start];
              for (uint64_t offset = tid; offset < nElem4; offset += blockDim.x) {
                data4[offset].x += scratch4[offset].x;
                data4[offset].y += scratch4[offset].y;
                data4[offset].z += scratch4[offset].z;
                data4[offset].w += scratch4[offset].w;
              }
              for (uint64_t offset = tid; offset < nLastElem; offset += blockDim.x) {
                data[d_start + nElem4 * 4 + offset] += recv_scratches[i][s_start + nElem4 * 4 + offset];
              }
              reduced[i] += diff;
              count[i] += diff;
            } while (reduced[i] < ready_loop);
            __syncthreads();
          }
          if (reduced[i] < rloop) rloop = reduced[i];
        }

        for (int i = tid; i < nrecv_sm + nrecv_proxy; i += blockDim.x) {
          if (count[i] > 0) {
            if (i < nrecv_sm) recv_sm_channels[i].signal(count[i]);
            else {
              const int before = reduced[i] - count[i];
              if (before > 0 && before / FLUSH_INTERVAL < reduced[i] / FLUSH_INTERVAL) recv_proxy_channels[i - nrecv_sm].flush();
              recv_proxy_channels[i - nrecv_sm].signal(count[i]);
            }
          }
        }
        __syncthreads(); // Necessary; otherwise, program can freeze in multirun allreduce.
      }

      if (nsend_proxy == 1) {
        int psends = pending_sends;
        if (psends == 0) {
          poll_loop_cnt = 0;
        } else {
          ++poll_loop_cnt;
          if (poll_loop_cnt >= 10) {
            if (tid == 0) pending_sends = psends - send_proxy_channels[0].poll(psends);
            poll_loop_cnt = 0;
            __syncthreads();
          }
        }
      }

      if (sloop < rloop) {
        // assert nsend_sm + nsend_proxy > 0
        if (node_type == 0) { // root
          for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].signal(rloop - sloop);
          for (int loop = sloop; loop < rloop; ++loop) {
            const uint64_t d_start = data_start + loop * nelem_per_send;
            const uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
            for (int i = tid; i < nsend_proxy; i += blockDim.x) {
              if (loop == 0) send_proxy_channels[i].wait();
              else if (loop % FLUSH_INTERVAL == 0) send_proxy_channels[i].flush();
              send_proxy_channels[i].putWithSignal(d_start * sizeof(int), size * sizeof(int));
            }
          }
          sloop = rloop;
        } else {
          // assert nsend_sm + nsend_proxy == 1
          int psends = pending_sends;
          if (psends == max_pending_sends) {
            if (tid == 0) pending_sends = psends - (nsend_sm == 1 ? send_sm_channels[0].poll(psends) : 
                                                                    send_proxy_channels[0].poll(psends));
            poll_loop_cnt = 0;
            __syncthreads();
            psends = pending_sends;
            if (psends == max_pending_sends) {
              __syncthreads();
              continue;
            }
          }
          
          // pipeline send: ensure one send (one nelem_per_send) one signal
          if (nsend_sm == 1) {
            do {
              uint64_t s_start = (sloop % max_pending_sends) * nelem_per_send;
              uint64_t d_start = data_start + sloop * nelem_per_send;
              uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
              send_sm_channels[0].put(s_start * sizeof(int), d_start * sizeof(int), size * sizeof(int), tid, blockDim.x);
              ++sloop;
              ++psends;
              __syncthreads();
              if (tid == 0) send_sm_channels[0].signal();
            } while (psends < max_pending_sends && sloop < rloop);
          } else { // nsend_proxy == 1
            do {
              uint64_t s_start = (sloop % max_pending_sends) * nelem_per_send;
              uint64_t d_start = data_start + sloop * nelem_per_send;
              uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
              if (tid == 0) {
                if (sloop > 0 && sloop % FLUSH_INTERVAL == 0) send_proxy_channels[0].flush();
                send_proxy_channels[0].putWithSignal(s_start * sizeof(int), d_start * sizeof(int), size * sizeof(int));
              }
              ++sloop;
              ++psends;
            } while (psends < max_pending_sends && sloop < rloop);
          }
          if (tid == 0) pending_sends = psends;
          __syncthreads();
        }
      }
    }
    if (node_type == 0) { // root
      for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].wait();
    } else {
      if (tid == 0 && nsend_sm + nsend_proxy == 1) {
        int psends = pending_sends;
        do {
          psends -= (nsend_sm == 1 ? send_sm_channels[0].poll(psends) : 
                                     send_proxy_channels[0].poll(psends));
        } while (psends > 0);
      }
    }
  } else {
    // assert nrecv_sm + nrecv_proxy <= 1
    if (nrecv_sm == 0 && nrecv_proxy == 0) {
      for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].signal(nloops);
      for (int sloop = 0; sloop < nloops; ++sloop) {
        const uint64_t d_start = data_start + sloop * nelem_per_send;
        const uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
        for (int i = tid; i < nsend_proxy; i += blockDim.x) {
          if (sloop == 0) send_proxy_channels[i].wait();
          else send_proxy_channels[i].flush();
          send_proxy_channels[i].putWithSignal(d_start * sizeof(int), size * sizeof(int));
        }
      }
    } else {
      if (tid == 0 && nrecv_proxy == 1) recv_proxy_channels[0].signal();
      int sloop = 0;
      __shared__ int ready;
      if (tid == 0) ready = 0;
      while (sloop < nloops) {
        if (tid == 0) {
          int ready_loop = sloop;
          if (nrecv_sm == 1) {
            do {
              ready_loop += recv_sm_channels[0].poll(nloops - ready_loop);
            } while (ready_loop == sloop);
          } else {
            do {
              ready_loop += recv_proxy_channels[0].poll(nloops - ready_loop);
            } while (ready_loop == sloop);
          }
          ready = ready_loop;
        }
        __syncthreads();
        const int ready_loop = ready;
        do {
          uint64_t d_start = data_start + sloop * nelem_per_send;
          uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
          if (nrecv_sm == 1) recv_sm_channels[0].get(d_start * sizeof(int), size * sizeof(int), tid, blockDim.x);
          __syncthreads();
          for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].signal();
          for (int i = tid; i < nsend_proxy; i += blockDim.x) {
            if (sloop == 0) send_proxy_channels[i].wait();
            else if (sloop % FLUSH_INTERVAL == 0) send_proxy_channels[i].flush();
            send_proxy_channels[i].putWithSignal(d_start * sizeof(int), size * sizeof(int));
          }
          ++sloop;
        } while (sloop < ready_loop);
      }
    }
    if (tid == 0 && nrecv_sm == 1) recv_sm_channels[0].signal(); // `signal` to ensure sender wait until `get` finishes
    for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].wait();
  }
  for (int i = tid; i < nrecv_proxy; i += blockDim.x) recv_proxy_channels[i].flush();
  for (int i = tid; i < nsend_proxy; i += blockDim.x) send_proxy_channels[i].flush();
}

/// Call threadblockCall.
/// SM channel scratches: recv_scratches[block_scratch_starts[bid], ..., block_scratch_starts[bid] + nrecvs_sm[bid] - 1]
/// Proxy channel scratches: recv_scratches[block_scratch_starts[bid] + nrecvs_sm[bid], ..., block_scratch_starts[bid] + nrecvs_sm[bid] + nrecvs_proxy[bid] - 1]
extern "C" __global__ void __launch_bounds__(1024)
    KERNEL(mscclpp::SmChannelDeviceHandle* recv_sm_channels, mscclpp::SmChannelDeviceHandle* send_sm_channels,
           mscclpp::SimpleProxyChannelDeviceHandle* recv_proxy_channels, mscclpp::SimpleProxyChannelDeviceHandle* send_proxy_channels,
           int** recv_scratches, int* block_recv_sm_ch_starts, int* block_send_sm_ch_starts,
           int* block_recv_proxy_ch_starts, int* block_send_proxy_ch_starts,
           int* block_scratch_starts,
           int* nrecvs_sm, int* nsends_sm, int* nrecvs_proxy, int* nsends_proxy,
           char* node_types, const uint64_t scratch_size, int* data, 
           const uint64_t* data_start, const uint64_t nelem_per_send, const uint64_t* nelem_total) {
  const int bid = blockIdx.x;

  threadblockCall(recv_sm_channels == nullptr ? nullptr : &recv_sm_channels[block_recv_sm_ch_starts[bid]], 
                  send_sm_channels == nullptr ? nullptr : &send_sm_channels[block_send_sm_ch_starts[bid]],
                  recv_proxy_channels == nullptr ? nullptr : &recv_proxy_channels[block_recv_proxy_ch_starts[bid]], 
                  send_proxy_channels == nullptr ? nullptr : &send_proxy_channels[block_send_proxy_ch_starts[bid]],
                  &recv_scratches[block_scratch_starts[bid]], nrecvs_sm[bid], nsends_sm[bid], nrecvs_proxy[bid], nsends_proxy[bid],
                  node_types[bid], scratch_size, data,
                  data_start[bid], nelem_per_send, nelem_total[bid]);
}