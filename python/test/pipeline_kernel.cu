#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <mscclpp/sm_channel_device.hpp>
#include <mscclpp/proxy_channel_device.hpp>

// BEGIN_DEFINES //

#ifndef PARAMETRIZE
#define KERNEL pipeline_schedule
#define N_PEERS 8
#endif

// END_DEFINES //

/// The call is a single node in the tree.
///
/// @param recv_sm_channels SM channels for recv.
/// @param send_sm_channels SM channels for send.
/// @param recv_proxy_channels Proxy channels for recv.
/// @param send_proxy_channels Proxy channels for send.
/// @param recv_scratches Scratch buffers for each recv_sm_channels, len(recv_scratches) == len(recv_sm_channels) + len(recv_proxy_channels)
/// @param nrecv_sm Num of valid recv_sm_channels (nrecv_sm + nrecv_proxy <= 1 if broadcast).
/// @param nsend_sm Num of valid send_sm_channels (nsend_sm + nsend_proxy <= 1 if reduce).
/// @param nrecv_proxy Num of valid recv_proxy_channels (nrecv_sm + nrecv_proxy <= 1 if broadcast).
/// @param nsend_proxy Num of valid send_proxy_channels (nsend_sm + nsend_proxy <= 1 if reduce).
/// @param node_type <0: reduce node; =0: root node; >0: broadcast node.
/// The send channels of broadcast and root nodes write to data buffer.
/// The send channels of reduce node write to scratch buffer.
/// @param data_start The data buffer start.
/// @param nelem_per_send Num of elements in each send.
/// @param nelem_total Total num of elements need to be send/recv.
/// @param scratch_size Max num of elements in scratch buffer for each recv channel (ignore if not reduce).
/// scratch_size must be greater than nelem_per_send
/// @param data Data buffer.
MSCCLPP_DEVICE_INLINE void 
    threadblockCall(mscclpp::SmChannelDeviceHandle* recv_sm_channels, mscclpp::SmChannelDeviceHandle* send_sm_channels,
                    mscclpp::SimpleProxyChannelDeviceHandle* recv_proxy_channels, mscclpp::SimpleProxyChannelDeviceHandle* send_proxy_channels,
                    int** recv_scratches, const int nrecv_sm, const int nsend_sm, const int nrecv_proxy, const int nsend_proxy,
                    const char node_type, const uint64_t data_start,
                    const uint64_t nelem_per_send, const uint64_t nelem_total, const uint64_t scratch_size, int* data) {
  const int tid = threadIdx.x;

  const int nloops = (nelem_total + nelem_per_send - 1) / nelem_per_send; // ceiling division

  if (node_type <= 0) {
    int reduced[N_PEERS] = {};
    __shared__ int ready[N_PEERS];
    __shared__ int pending_sends; // Only reduce node needs to track pending sends.
                                  // Reduce node has at most one send peer.

    #pragma unroll
    for (int i = tid; i < N_PEERS; i += blockDim.x) ready[i] = 0;
    if (tid == 0) pending_sends = 0;
    __syncthreads();

    const int max_pending_sends = scratch_size / nelem_per_send;

    int rloop = nrecv_sm + nrecv_proxy > 0 ? 0 : nloops; // progress of recv
    int sloop = nsend_sm + nsend_proxy > 0 ? 0 : nloops; // progress of send
    while (rloop < nloops || sloop < nloops) {
      if (rloop < nloops) {
        // assert nrecv_sm + nrecv_proxy > 0
        for (int i = tid; i < nrecv_sm + nrecv_proxy; i += blockDim.x) {
          const int ready_loop = ready[i];
          // if (ready_loop < rloop + 1) ready[i] += (i < nrecv_sm ? recv_sm_channels[i].poll(rloop + 1 - ready_loop) :
          //                                                         recv_proxy_channels[i - nrecv_sm].poll(rloop + 1 - ready_loop));
          if (ready_loop < nloops) ready[i] += (i < nrecv_sm ? recv_sm_channels[i].poll(nloops - ready_loop) :
                                                               recv_proxy_channels[i - nrecv_sm].poll(nloops - ready_loop));
        }
        __syncthreads();

        int count[N_PEERS] = {};
        rloop = nloops;
        for (int i = 0; i < nrecv_sm + nrecv_proxy; ++i) {
          const int ready_loop = ready[i];
          if (reduced[i] < ready_loop){
            do {
              const uint64_t s_start = (reduced[i] % max_pending_sends) * nelem_per_send;
              const uint64_t d_start = data_start + reduced[i] * nelem_per_send;
              const int diff = min(ready_loop - reduced[i], max_pending_sends - reduced[i] % max_pending_sends);

              const uint64_t nElem = min(nelem_per_send * diff, data_start + nelem_total - d_start);
              const uint64_t nElem4 = nElem / 4;
              const uint64_t nLastElem = nElem % 4;

              int4* const data4 = (int4*) &data[d_start];
              int4* const scratch4 = (int4*) &recv_scratches[i][s_start];
              for (uint64_t offset = tid; offset < nElem4; offset += blockDim.x) {
                data4[offset].w += scratch4[offset].w;
                data4[offset].x += scratch4[offset].x;
                data4[offset].y += scratch4[offset].y;
                data4[offset].z += scratch4[offset].z;
              }
              for (uint64_t offset = tid; offset < nLastElem; offset += blockDim.x) {
                data[d_start + nElem4 * 4 + offset] += recv_scratches[i][s_start + nElem4 * 4 + offset];
              }
              reduced[i] += diff;
              count[i] += diff;
            } while (reduced[i] < ready_loop);
            __syncthreads();
          }
          if (reduced[i] < rloop) rloop = reduced[i];
        }

        for (int i = tid; i < nrecv_sm + nrecv_proxy; i += blockDim.x) {
          if (count[i] > 0) {
            if (i < nrecv_sm) recv_sm_channels[i].signal(count[i]);
            else recv_proxy_channels[i - nrecv_sm].signal(count[i]);
          }
        }
      }

      if (sloop < rloop) {
        // assert nsend_sm + nsend_proxy > 0
        if (node_type == 0) { // root
          for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].signal(rloop - sloop);
          for (int i = tid; i < nsend_proxy; i += blockDim.x) {
            for (int loop = sloop; loop < rloop; ++loop) {
              uint64_t d_start = data_start + loop * nelem_per_send;
              uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
              send_proxy_channels[i].putWithSignal(d_start * sizeof(int), size * sizeof(int));
            }
          }
          sloop = rloop;
        } else {
          // assert nsend_sm + nsend_proxy == 1
          int psends = pending_sends;
          if (psends == max_pending_sends) {
            if (tid == 0) pending_sends -= (nsend_sm == 1 ? send_sm_channels[0].poll(psends) : 
                                                            send_proxy_channels[0].poll(psends));
            __syncthreads();
            psends = pending_sends;
            if (psends == max_pending_sends) {
              __syncthreads();
              continue;
            }
          }
          
          // pipeline send: ensure one send (one nelem_per_send) one signal
          if (nsend_sm == 1) {
            do {
              uint64_t s_start = (sloop % max_pending_sends) * nelem_per_send;
              uint64_t d_start = data_start + sloop * nelem_per_send;
              uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
              send_sm_channels[0].put(s_start * sizeof(int), d_start * sizeof(int), size * sizeof(int), tid, blockDim.x);
              ++sloop;
              ++psends;
              __syncthreads();
              if (tid == 0) send_sm_channels[0].signal();
            } while (psends < max_pending_sends && sloop < rloop);
          } else { // nsend_proxy == 1
            do {
              uint64_t s_start = (sloop % max_pending_sends) * nelem_per_send;
              uint64_t d_start = data_start + sloop * nelem_per_send;
              uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
              if (tid == 0) send_proxy_channels[0].putWithSignal(s_start * sizeof(int), d_start * sizeof(int), size * sizeof(int));
              ++sloop;
              ++psends;
            } while (psends < max_pending_sends && sloop < rloop);
          }
          if (tid == 0) pending_sends = psends;
          __syncthreads();
        }
      }
    }
  } else {
    // assert nrecv_sm + nrecv_proxy <= 1
    if (nrecv_sm == 0 && nrecv_proxy == 0) {
      for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].signal(nloops);
      for (int i = tid; i < nsend_proxy; i += blockDim.x) {
        for (int sloop = 0; sloop < nloops; ++sloop) {
          uint64_t d_start = data_start + sloop * nelem_per_send;
          uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
          send_proxy_channels[i].putWithSignal(d_start * sizeof(int), size * sizeof(int));
        }
      }
    } else {
      int sloop = 0;
      __shared__ int ready;
      if (tid == 0) ready = 0;
      while (sloop < nloops) {
        if (tid == 0) {
          int ready_loop = sloop;
          if (nrecv_sm == 1) {
            do {
              ready_loop += recv_sm_channels[0].poll(nloops - ready_loop);
            } while (ready_loop == sloop);
          } else {
            do {
              ready_loop += recv_proxy_channels[0].poll(nloops - ready_loop);
            } while (ready_loop == sloop);
          }
          ready = ready_loop;
        }
        __syncthreads();
        const int ready_loop = ready;
        do {
          uint64_t d_start = data_start + sloop * nelem_per_send;
          uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
          for (int i = tid; i < nsend_proxy; i += blockDim.x) {
            send_proxy_channels[i].putWithSignal(d_start * sizeof(int), size * sizeof(int));
          }
          if (nrecv_sm == 1) recv_sm_channels[0].get(d_start * sizeof(int), size * sizeof(int), tid, blockDim.x);
          ++sloop;
          __syncthreads();
          for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].signal();
        } while (sloop < ready_loop);
      }
    }
  }
  for (int i = tid; i < nsend_proxy; i += blockDim.x) send_proxy_channels[i].flush(); // question?
}

/// Call threadblockCall.
/// SM channel scratches: recv_scratches[block_scratch_starts[bid], ..., block_scratch_starts[bid] + nrecvs_sm[bid] - 1]
/// Proxy channel scratches: recv_scratches[block_scratch_starts[bid] + nrecvs_sm[bid], ..., block_scratch_starts[bid] + nrecvs_sm[bid] + nrecvs_proxy[bid] - 1]
extern "C" __global__ void __launch_bounds__(1024)
    KERNEL(mscclpp::SmChannelDeviceHandle* recv_sm_channels, mscclpp::SmChannelDeviceHandle* send_sm_channels,
           mscclpp::SimpleProxyChannelDeviceHandle* recv_proxy_channels, mscclpp::SimpleProxyChannelDeviceHandle* send_proxy_channels,
           int** recv_scratches, int* block_recv_sm_ch_starts, int* block_send_sm_ch_starts,
           int* block_recv_proxy_ch_starts, int* block_send_proxy_ch_starts,
           int* block_scratch_starts,
           int* nrecvs_sm, int* nsends_sm, int* nrecvs_proxy, int* nsends_proxy,
           char* node_types, uint64_t* data_start, const uint64_t nelem_per_send,
           uint64_t* nelem_total, const uint64_t scratch_size, int* data) {
  const int bid = blockIdx.x;

  threadblockCall(recv_sm_channels == nullptr ? nullptr : &recv_sm_channels[block_recv_sm_ch_starts[bid]], 
                  send_sm_channels == nullptr ? nullptr : &send_sm_channels[block_send_sm_ch_starts[bid]],
                  recv_proxy_channels == nullptr ? nullptr : &recv_proxy_channels[block_recv_proxy_ch_starts[bid]], 
                  send_proxy_channels == nullptr ? nullptr : &send_proxy_channels[block_send_proxy_ch_starts[bid]],
                  &recv_scratches[block_scratch_starts[bid]], nrecvs_sm[bid], nsends_sm[bid], nrecvs_proxy[bid], nsends_proxy[bid],
                  node_types[bid], data_start[bid], nelem_per_send, nelem_total[bid], scratch_size, data);
}