#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <mscclpp/concurrency_device.hpp>
#include <mscclpp/sm_channel_device.hpp>
#include <mscclpp/proxy_channel_device.hpp>
#include <assert.h>

// BEGIN_DEFINES //

#ifndef PARAMETRIZE
#define KERNEL pipeline_reduceScatter_schedule
#endif

#define FLUSH_INTERVAL 50
#define MAX_NLOOPS 1048576

// END_DEFINES //


MSCCLPP_DEVICE_INLINE void
    threadblockCall(mscclpp::SmChannelDeviceHandle* recv_sm_channel, mscclpp::SmChannelDeviceHandle* send_sm_channel,
                    mscclpp::SimpleProxyChannelDeviceHandle* recv_proxy_channel, mscclpp::SimpleProxyChannelDeviceHandle* send_proxy_channel,
                    int* recv_scratch, const bool recv_sm, const bool send_sm, const bool recv_proxy, const bool send_proxy,
                    const uint64_t scratch_size, int* data,
                    int* reduce_locks, int* reduce_counts, const int nrecv_peers, int* sent_progress,
                    const uint64_t data_start, const uint64_t nelem_per_send, const uint64_t nelem_total) {
  const int tid = threadIdx.x;

  const int nloops = (nelem_total + nelem_per_send - 1) / nelem_per_send; // ceiling division
  assert(nloops <= MAX_NLOOPS);

  int pending_sends = 0;
  const int max_pending_sends = scratch_size / nelem_per_send;

  if (tid == 0) {
    assert(reinterpret_cast<uintptr_t>(data) % alignof(int4) == 0);
    assert(data_start % 4 == 0);
    assert(nelem_per_send % 4 == 0);
    assert((recv_sm || recv_proxy) == (nrecv_peers > 0));
    assert(!send_sm || !send_proxy);
    assert(!recv_sm || !recv_proxy);
  }

  __shared__ int ready;
  __shared__ int sent;

  if (tid == 0) {
    ready = (recv_sm || recv_proxy ? 0 : nloops);
    sent = (recv_sm || send_proxy ? 0 : nloops);

    if (!recv_sm && !recv_proxy && send_sm) {
      send_sm_channel->signal(nloops);
      sent = nloops;
    }
  }
  __syncthreads();

  int received = (recv_sm || recv_proxy ? 0 : nloops);
  int reduced = (recv_sm || recv_proxy ? 0 : nloops);

  while (reduced < nloops || sent < nloops) {
    if (received < nloops) {
      int ready_local = ready;
      if (ready_local == received) {
        if (tid == 0) {
          if (recv_sm) {
            ready_local += recv_sm_channel->poll(nloops - ready);
          } else {
            ready_local += recv_proxy_channel->poll(nloops - ready);
          }
          ready = ready_local;
        }
        __syncthreads();
        ready_local = ready;
      }
      if (ready_local > received) {
        if (recv_sm && nrecv_peers > 1) {
          // only recv sm channel with no rrcs use get
          // recv sm channel with rrcs will directly use read when reduce
          // proxy channel does not have get
          const uint64_t s_start = (received % max_pending_sends) * nelem_per_send;
          const uint64_t d_start = data_start + received * nelem_per_send;
          const uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
          recv_sm_channel->get(s_start * sizeof(int), d_start * sizeof(int), size * sizeof(int), tid, blockDim.x);
          ++received;
        } else {
          received = ready_local;
        }
      }
      __syncthreads();
    }

    if (reduced < received) {
      const uint64_t s_start = (reduced % max_pending_sends) * nelem_per_send;
      const uint64_t d_start = data_start + reduced * nelem_per_send;
      const uint64_t d_start4 = d_start / 4;
      const uint64_t nElem = min(nelem_per_send, data_start + nelem_total - d_start);
      const uint64_t nElem4 = nElem / 4;
      const uint64_t nLastElem = nElem % 4;
      int4* const data4 = reinterpret_cast<int4*>(&data[d_start]);
      bool ready_to_send_sm = false; // only needed by thread 0
      if (recv_sm && nrecv_peers == 1) {
        // no __threadfence() needed, only one threadblock is writing
        for (uint64_t offset = tid; offset < nElem4; offset += blockDim.x) {
          int4 tmp = data4[offset];
          int4 val = recv_sm_channel->read<int4>(d_start4 + offset);
          tmp.w += val.w;
          tmp.x += val.x;
          tmp.y += val.y;
          tmp.z += val.z;
          data4[offset] = tmp;
        }
        if (nLastElem > 0 && tid == 0) {
          int4 tmp = data4[nElem4];
          int4 val = recv_sm_channel.read<int4>(d_start4 + nElem4);
          // assert 1 <= nLastElem <= 3
          tmp.w += val.w;
          if (nLastElem > 1) tmp.x += val.x;
          if (nLastElem > 2) tmp.y += val.y;
          data4[nElem4] = tmp;
        }
        ready_to_send_sm = true;
        ++reduced;
      } else {
        // Try lock
        __shared__ int lock_status;
        if (tid == 0) lock_status = atomicCAS(&reduce_locks[reduced], 0, 1);
        __syncthreads();
        if (!lock_status) {
          __threadfence();
          int4* const scratch4 = reinterpret_cast<int4*>(&recv_scratch[s_start]);
          for (uint64_t offset = tid; offset < nElem4; offset += blockDim.x) {
            int4 tmp = data4[offset];
            int4 val = scratch4[offset];
            tmp.w += val.w;
            tmp.x += val.x;
            tmp.y += val.y;
            tmp.z += val.z;
            data4[offset] = tmp;
          }
          if (nLastElem > 0 && tid == 0) {
            int4 tmp = data4[nElem4];
            int4 val = scratch4[nElem4];
            // assert 1 <= nLastElem <= 3
            tmp.w += val.w;
            if (nLastElem > 1) tmp.x += val.x;
            if (nLastElem > 2) tmp.y += val.y;
            data4[nElem4] = tmp;
          }

          // Unlock
          __syncthreads();
          if (tid == 0) {
            if (++reduce_counts[reduced] == nrecv_peers) ready_to_send_sm = true;
            reduce_locks[reduced] = 0;
            if (recv_proxy) {
              if (reduced > 0 && reduced % FLUSH_INTERVAL == 0) recv_proxy_channel->flush();
              recv_proxy_channel->signal();
            }
          }
          ++reduced;
        }
      }
      if (tid == 0 && ready_to_send_sm && send_sm) {
        send_sm_channel->signal();
        ++sent;
      }
      __syncthreads();
    }

    if (send_proxy) {
      if (tid == 0) {
        int sent_local = sent;
        if (sent_local < reduced) {
          bool ready_to_send_proxy = false;
          if (nrecv_peers == 1) {
            ready_to_send_proxy = true;
          } else {
            do {
              if (reduce_counts[sent_local] == nrecv_peers) {
                const int global_sent = atomicCAS(sent_progress, sent_local, sent_local + 1);
                if (global_sent == sent_local) {
                  ready_to_send_proxy = true;
                } else {
                  sent_local = global_sent;
                }
              }
            } while (!ready_to_send_proxy && sent_local < reduced);
          }
          if (ready_to_send_proxy) {
            if (pending_sends == max_pending_sends) {
              pending_sends -= send_proxy_channel->poll(pending_sends);
            }
            if (pending_sends < max_pending_sends) {
              const int64_t s_start = (sent_local % max_pending_sends) * nelem_per_send;
              const uint64_t d_start = data_start + sent_local * nelem_per_send;
              const uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
              if (sent_local > 0 && sent_local % FLUSH_INTERVAL == 0) send_proxy_channel->flush();
              send_proxy_channel->putWithSignal(s_start * sizeof(int), d_start * sizeof(int), size * sizeof(int));
              ++pending_sends;
              ++sent_local;
            }
          }
          sent = sent_local;
        }
      }
      __syncthreads();
    }
  }

}

__device__ mscclpp::DeviceSyncer deviceSyncer;

MSCCLPP_DEVICE_INLINE void zero_memory(int* data, const uint64_t nelem) {
  const int tid = threadIdx.x;
  int4* data4 = reinterpret_cast<int4*>(data);
  const uint64_t nElem4 = nElem / 4;
  const uint64_t nLastElem = nElem % 4;
  for (uint64_t off = tid; off < nElem4; off += blockDim.x) {
    int4 tmp = data4[offset];
    tmp.w = 0;
    tmp.x = 0;
    tmp.y = 0;
    tmp.z = 0;
    data4[offset] = tmp;
  }
  if (nLastElem > 0 && tid == 0) {
    int4 tmp = data4[nElem4];
    tmp.w = 0;
    if (nLastElem > 1) tmp.x = 0;
    if (nLastElem > 2) tmp.y = 0;
    data4[nElem4] = tmp;
  }
}

extern "C" __global__ void __launch_bounds__(1024)
    KERNEL(mscclpp::SmChannelDeviceHandle* recv_sm_channel_block, mscclpp::SmChannelDeviceHandle* send_sm_channel_block,
           mscclpp::SimpleProxyChannelDeviceHandle* recv_proxy_channel_block, mscclpp::SimpleProxyChannelDeviceHandle* send_proxy_channel_block,
           int* recv_sm_channel_indics, int* send_sm_channel_indics, int* recv_proxy_channel_indics, int* send_proxy_channel_indics,
           int** recv_scratch_block, const uint64_t scratch_size, int* data,
           int** reduce_locks_block, int** reduce_counts_block, int* nrecv_peers_block, int** sent_progress_block, bool* first_block,
           const uint64_t* data_start_block, const uint64_t nelem_per_send, const uint64_t* nelem_total_block) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const bool is_first_block = first_block[bid];
  mscclpp::SmChannelDeviceHandle* recv_sm_channel = (recv_sm_channel_indics[bid] < 0 ? nullptr : &recv_sm_channel_block[recv_sm_channel_indics[bid]]);
  mscclpp::SmChannelDeviceHandle* send_sm_channel = (send_sm_channel_indics[bid] < 0 ? nullptr : &send_sm_channel_block[send_sm_channel_indics[bid]]);
  mscclpp::SimpleProxyChannelDeviceHandle* recv_proxy_channel = (recv_proxy_channel_indics[bid] < 0 ? nullptr : &recv_proxy_channel_block[recv_proxy_channel_indics[bid]]);
  mscclpp::SimpleProxyChannelDeviceHandle* send_proxy_channel = (send_proxy_channel_indics[bid] < 0 ? nullptr : &send_proxy_channel_block[send_proxy_channel_indics[bid]]);
  int* recv_scratch = recv_scratch_block[bid];
  const bool recv_sm = (recv_sm_channel != nullptr);
  const bool send_sm = (send_sm_channel != nullptr);
  const bool recv_proxy = (recv_proxy_channel != nullptr);
  const bool send_proxy = (send_proxy_channel != nullptr);
  int* reduce_locks = reduce_locks_block[bid];
  int* reduce_counts = reduce_counts_block[bid];
  const int nrecv_peers = nrecv_peers_block[bid];
  int* sent_progress = sent_progress_block[bid];
  const uint64_t data_start = data_start_block[bid];
  const uint64_t nelem_total = nelem_total_block[bid];

  if (is_first_block) {
    if (recv_sm || recv_proxy) {
      *sent_progress = 0;
      const int nloops = (nelem_total + nelem_per_send - 1) / nelem_per_send;
      zero_memory(reduce_locks, nloops);
      zero_memory(reduce_counts, nloops);
    }
  }
  deviceSyncer.sync(gridDim.x);


  threadblockCall(recv_sm_channel, send_sm_channel, recv_proxy_channel, send_proxy_channel,
                  recv_scratch, recv_sm, send_sm, recv_proxy, send_proxy,
                  scratch_size, data,
                  reduce_locks, reduce_counts, nrecv_peers, sent_progress,
                  data_start, nelem_per_send, nelem_total);
}