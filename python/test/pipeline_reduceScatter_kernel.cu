#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <mscclpp/concurrency_device.hpp>
#include <mscclpp/sm_channel_device.hpp>
#include <mscclpp/proxy_channel_device.hpp>
#include <assert.h>

// BEGIN_DEFINES //

#ifndef PARAMETRIZE
#define KERNEL pipeline_reduceScatter_schedule
#endif

#define FLUSH_INTERVAL 50
#define MAX_NLOOPS 1048576

// END_DEFINES //


MSCCLPP_DEVICE_INLINE void
    threadblockCall(mscclpp::SmChannelDeviceHandle* recv_sm_channel, mscclpp::SmChannelDeviceHandle* send_sm_channel,
                    mscclpp::SimpleProxyChannelDeviceHandle* recv_proxy_channel, mscclpp::SimpleProxyChannelDeviceHandle* send_proxy_channel,
                    int* recv_scratch, const bool recv_sm, const bool send_sm, const bool recv_proxy, const bool send_proxy,
                    const uint64_t scratch_size, int* data,
                    int* reduce_counts, const int nrecv_peers, const bool is_first_block,
                    const uint64_t data_start, const uint64_t nelem_per_send, const uint64_t nelem_total, const uint64_t debug_flag) {
  const int tid = threadIdx.x;

  const int nloops = (nelem_total + nelem_per_send - 1) / nelem_per_send; // ceiling division
  assert(nloops <= MAX_NLOOPS);

  int pending_sends = 0; // only thread 0 at is_first_block needs this
  const int max_pending_sends = scratch_size / nelem_per_send;

  if (tid == 0) {
    assert(reinterpret_cast<uintptr_t>(data) % alignof(int4) == 0);
    assert(data_start % 4 == 0);
    assert(nelem_per_send % 4 == 0);
    assert((recv_sm || recv_proxy) == (nrecv_peers > 0));
    assert(!send_sm || !send_proxy);
    assert(!recv_sm || !recv_proxy);
  }

  int received = (recv_sm || recv_proxy ? 0 : nloops);
  int reduced = (recv_sm || recv_proxy ? 0 : nloops);

  int sent_local;
  if (is_first_block) {
    if (!recv_sm && !recv_proxy && send_sm) {
      if (tid == 0) send_sm_channel->signal(nloops);
      sent_local = nloops;
    } else {
      sent_local = (send_sm || send_proxy ? 0 : nloops);
    }
  } else {
    sent_local = nloops;
  }

  __shared__ int ready;
  __shared__ int sent;
  if (tid == 0) {
    ready = (recv_sm || recv_proxy ? 0 : nloops);
    sent = sent_local;
  }
  __syncthreads();

  while (reduced < nloops || sent_local < nloops) {
    if (received < nloops) {
      // assert recv_sm or recv_proxy
      int ready_local = ready;
      if (ready_local == received) {
        if (tid == 0) {
          if (recv_sm) {
            ready_local += recv_sm_channel->poll(nloops - ready);
          } else {
            ready_local += recv_proxy_channel->poll(nloops - ready);
          }
          ready = ready_local;
        }
        __syncthreads();
        ready_local = ready;
      }
      if (ready_local > received) {
        if (recv_sm && nrecv_peers > 1) {
          // only recv sm channel with no rrcs use get
          // recv sm channel with rrcs will directly use read when reduce
          // proxy channel does not have get
          const uint64_t s_start = (received % max_pending_sends) * nelem_per_send;
          const uint64_t d_start = data_start + received * nelem_per_send;
          const uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
          recv_sm_channel->get(d_start * sizeof(int), s_start * sizeof(int), size * sizeof(int), tid, blockDim.x);
          ++received;
        } else {
          received = ready_local;
        }
      }
      __syncthreads();
    }

    if (reduced < received) {
      const uint64_t s_start = (reduced % max_pending_sends) * nelem_per_send;
      const uint64_t d_start = data_start + reduced * nelem_per_send;
      const uint64_t d_start4 = d_start / 4;
      const uint64_t nElem = min(nelem_per_send, data_start + nelem_total - d_start);
      const uint64_t nElem4 = nElem / 4;
      const uint64_t nLastElem = nElem % 4;
      int4* const data4 = reinterpret_cast<int4*>(&data[d_start]);
      if (recv_sm && nrecv_peers == 1) {
        // no __threadfence() needed, only one threadblock is writing
        for (uint64_t offset = tid; offset < nElem4; offset += blockDim.x) {
          int4 tmp = data4[offset];
          int4 val = recv_sm_channel->read<int4>(d_start4 + offset);
          tmp.x += val.x;
          tmp.y += val.y;
          tmp.z += val.z;
          tmp.w += val.w;
          data4[offset] = tmp;
        }
        if (nLastElem > 0 && tid == 0) {
          int4 tmp = data4[nElem4];
          int4 val = recv_sm_channel->read<int4>(d_start4 + nElem4);
          // assert 1 <= nLastElem <= 3
          tmp.x += val.x;
          if (nLastElem > 1) tmp.y += val.y;
          if (nLastElem > 2) tmp.z += val.z;
          data4[nElem4] = tmp;
        }
        ++reduced;
      } else {
        // Try lock
        __shared__ int re_cnt;
        if (tid == 0) re_cnt = atomicExch(&reduce_counts[reduced], -1); // Try lock
        __syncthreads();
        if (re_cnt >= 0) {
          int4* const scratch4 = reinterpret_cast<int4*>(&recv_scratch[s_start]);
          for (uint64_t offset = tid; offset < nElem4; offset += blockDim.x) {
            int4 tmp = data4[offset];
            int4 val = scratch4[offset];
            tmp.x += val.x;
            tmp.y += val.y;
            tmp.z += val.z;
            tmp.w += val.w;
            data4[offset] = tmp;
          }
          if (nLastElem > 0 && tid == 0) {
            int4 tmp = data4[nElem4];
            int4 val = scratch4[nElem4];
            // assert 1 <= nLastElem <= 3
            tmp.x += val.x;
            if (nLastElem > 1) tmp.y += val.y;
            if (nLastElem > 2) tmp.z += val.z;
            data4[nElem4] = tmp;
          }

          // Unlock
          __syncthreads();
          if (tid == 0) {
            __threadfence();
            *((volatile int*) &reduce_counts[reduced]) = re_cnt + 1;
            if (recv_proxy) {
              if (reduced > 0 && reduced % FLUSH_INTERVAL == 0) recv_proxy_channel->flush();
              recv_proxy_channel->signal();
            }
          }
          ++reduced;
        }
      }
      __syncthreads();
    }

    if (sent_local < reduced) {
      // assert is_first_block
      // assert send_sm or send_proxy
      if (tid == 0) {
        if (nrecv_peers <= 1 || *((volatile int*) &reduce_counts[sent_local]) == nrecv_peers) {
          if (send_sm) {
            do {
              send_sm_channel->signal();
              ++sent_local;
            } while (sent_local < reduced && (nrecv_peers <= 1 || *((volatile int*) &reduce_counts[sent_local]) == nrecv_peers));
            sent = sent_local;
          } else {
            if (pending_sends == max_pending_sends) {
              pending_sends -= send_proxy_channel->poll(pending_sends);
            }
            if (pending_sends < max_pending_sends) {
              do {
                const uint64_t s_start = (sent_local % max_pending_sends) * nelem_per_send;
                const uint64_t d_start = data_start + sent_local * nelem_per_send;
                const uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
                if (sent_local > 0 && sent_local % FLUSH_INTERVAL == 0) send_proxy_channel->flush();
                send_proxy_channel->putWithSignal(s_start * sizeof(int), d_start * sizeof(int), size * sizeof(int));
                ++pending_sends;
                ++sent_local;
              } while (pending_sends < max_pending_sends && sent_local < reduced && 
                       (nrecv_peers <= 1 || *((volatile int*) &reduce_counts[sent_local]) == nrecv_peers));
              sent = sent_local;
            }
          }
        }
      }
      __syncthreads();
      sent_local = sent;
    }
    __syncthreads();
  }
  if (tid == 0) {
    if (recv_sm) recv_sm_channel->signal();
    if (send_sm && is_first_block) send_sm_channel->wait();
    if (recv_proxy) recv_proxy_channel->flush();
    if (send_proxy && is_first_block) {
      while (pending_sends > 0) pending_sends -= send_proxy_channel->poll(pending_sends);
      send_proxy_channel->flush();
    }
  }
  __syncthreads();
}

__device__ mscclpp::DeviceSyncer deviceSyncer;

MSCCLPP_DEVICE_INLINE void zero_memory(int* data, const uint64_t nelem) {
  const int tid = threadIdx.x;
  int4* data4 = reinterpret_cast<int4*>(data);
  const uint64_t nElem4 = nelem / 4;
  const uint64_t nLastElem = nelem % 4;
  for (uint64_t offset = tid; offset < nElem4; offset += blockDim.x) {
    int4 tmp = data4[offset];
    tmp.x = 0;
    tmp.y = 0;
    tmp.z = 0;
    tmp.w = 0;
    data4[offset] = tmp;
  }
  if (nLastElem > 0 && tid == 0) {
    int4 tmp = data4[nElem4];
    tmp.x = 0;
    if (nLastElem > 1) tmp.y = 0;
    if (nLastElem > 2) tmp.z = 0;
    data4[nElem4] = tmp;
  }
}

extern "C" __global__ void __launch_bounds__(1024)
    KERNEL(mscclpp::SmChannelDeviceHandle* recv_sm_channel_block, mscclpp::SmChannelDeviceHandle* send_sm_channel_block,
           mscclpp::SimpleProxyChannelDeviceHandle* recv_proxy_channel_block, mscclpp::SimpleProxyChannelDeviceHandle* send_proxy_channel_block,
           int* recv_sm_channel_indics, int* send_sm_channel_indics, int* recv_proxy_channel_indics, int* send_proxy_channel_indics,
           int** recv_scratch_block, const uint64_t scratch_size, int* data,
           int** reduce_counts_block, int* nrecv_peers_block, bool* first_block,
           const uint64_t* data_start_block, const uint64_t nelem_per_send, const uint64_t* nelem_total_block, const uint64_t debug_flag) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const bool is_first_block = first_block[bid];
  mscclpp::SmChannelDeviceHandle* recv_sm_channel = (recv_sm_channel_indics[bid] < 0 ? nullptr : &recv_sm_channel_block[recv_sm_channel_indics[bid]]);
  mscclpp::SmChannelDeviceHandle* send_sm_channel = (send_sm_channel_indics[bid] < 0 ? nullptr : &send_sm_channel_block[send_sm_channel_indics[bid]]);
  mscclpp::SimpleProxyChannelDeviceHandle* recv_proxy_channel = (recv_proxy_channel_indics[bid] < 0 ? nullptr : &recv_proxy_channel_block[recv_proxy_channel_indics[bid]]);
  mscclpp::SimpleProxyChannelDeviceHandle* send_proxy_channel = (send_proxy_channel_indics[bid] < 0 ? nullptr : &send_proxy_channel_block[send_proxy_channel_indics[bid]]);
  int* recv_scratch = recv_scratch_block[bid];
  const bool recv_sm = (recv_sm_channel != nullptr);
  const bool send_sm = (send_sm_channel != nullptr);
  const bool recv_proxy = (recv_proxy_channel != nullptr);
  const bool send_proxy = (send_proxy_channel != nullptr);
  int* reduce_counts = reduce_counts_block[bid];
  const int nrecv_peers = nrecv_peers_block[bid];
  const uint64_t data_start = data_start_block[bid];
  const uint64_t nelem_total = nelem_total_block[bid];

  if (is_first_block && (recv_sm || recv_proxy)) {
    const int nloops = (nelem_total + nelem_per_send - 1) / nelem_per_send;
    zero_memory(reduce_counts, nloops);
  }
  deviceSyncer.sync(gridDim.x);


  threadblockCall(recv_sm_channel, send_sm_channel, recv_proxy_channel, send_proxy_channel,
                  recv_scratch, recv_sm, send_sm, recv_proxy, send_proxy,
                  scratch_size, data,
                  reduce_counts, nrecv_peers, is_first_block,
                  data_start, nelem_per_send, nelem_total, debug_flag);
}