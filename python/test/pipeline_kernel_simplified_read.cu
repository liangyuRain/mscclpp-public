#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <mscclpp/sm_channel_device.hpp>
#include <mscclpp/proxy_channel_device.hpp>
#include <assert.h>

// BEGIN_DEFINES //

#ifndef PARAMETRIZE
#define KERNEL pipeline_schedule
#define N_PEERS 8
#endif

#define FLUSH_INTERVAL 100

// END_DEFINES //

/// The call is a single node in the tree.
///
/// Syncronization: It is guaranteed that in allreduce and allgather, after the kernel finishes,
/// the memory is safe to be written. However, the kernel immediately starts to write remote
/// memory once launched. There are two cases:
/// Allreduce/ReduceScatter - The kernel writes to remote scratch buffer in the beginning.
/// As long as the remote peer has finished initializing scratch buffer, the write is safe.
/// Allgather - The kernel (proxy) waits for remote signal before writing to remote.
///
/// @param recv_sm_channels SM channels for recv.
/// @param send_sm_channels SM channels for send.
/// @param recv_proxy_channels Proxy channels for recv.
/// @param send_proxy_channels Proxy channels for send.
/// @param recv_scratches Scratch buffers for each recv_sm_channels, len(recv_scratches) == len(recv_sm_channels) + len(recv_proxy_channels)
/// @param nrecv_sm Num of valid recv_sm_channels (nrecv_sm + nrecv_proxy <= 1 if broadcast).
/// @param nsend_sm Num of valid send_sm_channels (nsend_sm + nsend_proxy <= 1 if reduce).
/// @param nrecv_proxy Num of valid recv_proxy_channels (nrecv_sm + nrecv_proxy <= 1 if broadcast).
/// @param nsend_proxy Num of valid send_proxy_channels (nsend_sm + nsend_proxy <= 1 if reduce).
/// @param node_type <0: reduce node; =0: root node; >0: broadcast node.
/// The send channels of broadcast and root nodes write to data buffer.
/// The send channels of reduce node write to scratch buffer.
/// @param scratch_size Max num of elements in scratch buffer for each recv channel (ignore if not reduce).
/// scratch_size must be greater than nelem_per_send
/// @param data Data buffer.
/// @param data_start The data buffer start.
/// @param nelem_per_send Num of elements in each send.
/// @param nelem_total Total num of elements need to be send/recv.
MSCCLPP_DEVICE_INLINE void
    threadblockCall(mscclpp::SmChannelDeviceHandle* recv_sm_channels, mscclpp::SmChannelDeviceHandle* send_sm_channels,
                    mscclpp::SimpleProxyChannelDeviceHandle* recv_proxy_channels, mscclpp::SimpleProxyChannelDeviceHandle* send_proxy_channels,
                    int** recv_scratches, const int nrecv_sm, const int nsend_sm, const int nrecv_proxy, const int nsend_proxy,
                    const char node_type, const uint64_t scratch_size, int* data,
                    const uint64_t data_start, const uint64_t nelem_per_send, const uint64_t nelem_total) {
  const int tid = threadIdx.x;

  const int nloops = (nelem_total + nelem_per_send - 1) / nelem_per_send; // ceiling division

  if (node_type <= 0) {
    if (tid == 0) {
      assert(reinterpret_cast<uintptr_t>(data) % alignof(int4) == 0);
      assert(data_start % 4 == 0);
      assert(nelem_per_send % 4 == 0);
    }

    int reduced[N_PEERS] = {};
    __shared__ int ready[N_PEERS];
    int pending_sends = 0; // Only reduce node's proxy channel needs to track pending sends.
                           // Only thread(tid=0) needs to track pending sends.

    #pragma unroll
    for (int i = tid; i < N_PEERS; i += blockDim.x) ready[i] = 0;
    __syncthreads();

    const int max_pending_sends = scratch_size / nelem_per_send;

    int min_ready = 0;
    for (int loop = 0; loop < nloops; ++loop) {
      if (nrecv_sm + nrecv_proxy > 0) {
        const uint64_t s_start = (loop % max_pending_sends) * nelem_per_send;
        const uint64_t d_start = data_start + loop * nelem_per_send;
        const uint64_t d_start4 = d_start / 4;
        const uint64_t nElem = min(nelem_per_send, data_start + nelem_total - d_start);
        const uint64_t nElem4 = nElem / 4;
        const uint64_t nLastElem = nElem % 4;
        int4* const data4 = (int4*) &data[d_start];

        do {
          if (min_ready <= loop) {
            bool hasUpdate;
            do {
              hasUpdate = false;
              for (int i = tid; i < nrecv_sm + nrecv_proxy; i += blockDim.x) {
                const int ready_loop = ready[i];
                if (ready_loop < nloops) {
                  int update = (i < nrecv_sm ? recv_sm_channels[i].poll(nloops - ready_loop) :
                                               recv_proxy_channels[i - nrecv_sm].poll(nloops - ready_loop));
                  if (update > 0) hasUpdate = true;
                  ready[i] = ready_loop + update;
                }
              }
            } while (!__syncthreads_or(hasUpdate));
          }

          min_ready = nloops;
          bool chHasUpdate[N_PEERS];
          for (int i = 0; i < nrecv_sm + nrecv_proxy; ++i) {
            const int ready_loop = ready[i];
            if (ready_loop < min_ready) min_ready = ready_loop;
            if (ready_loop > loop && reduced[i] == loop) {
              chHasUpdate[i] = true;
              ++reduced[i];
            } else {
              chHasUpdate[i] = false;
            }
          }
          for (uint64_t offset = tid; offset < nElem4; offset += blockDim.x) {
            int4 tmp = data4[offset];
            for (int i = 0; i < nrecv_sm + nrecv_proxy; ++i) {
              if (chHasUpdate[i]) {
                int4 val;
                if (i < nrecv_sm) val = recv_sm_channels[i].read<int4>(d_start4 + offset);
                else val = reinterpret_cast<int4*>(&recv_scratches[i][s_start])[offset];
                tmp.w += val.w;
                tmp.x += val.x;
                tmp.y += val.y;
                tmp.z += val.z;
              }
            }
            data4[offset] = tmp;
          }
          if (nLastElem > 0 && tid == 0) {
            assert(loop == nloops - 1);
            int4 tmp = data4[nElem4];
            for (int i = 0; i < nrecv_sm + nrecv_proxy; ++i) {
              if (chHasUpdate[i]) {
                int4 val;
                if (i < nrecv_sm) val = recv_sm_channels[i].read<int4>(d_start4 + nElem4);
                else val = reinterpret_cast<int4*>(&recv_scratches[i][s_start])[nElem4];
                // assert 1 <= nLastElem <= 3
                tmp.w += val.w;
                if (nLastElem > 1) tmp.x += val.x;
                if (nLastElem > 2) tmp.y += val.y;
              }
            }
            data4[nElem4] = tmp;
          }
          __syncthreads();
          // sm_proxy_channels do not need to be signaled, because instead of remote writes to
          // local scratch buffer, we read from remote memory directly.
          for (int i = tid; i < nrecv_proxy; i += blockDim.x) {
            if (chHasUpdate[i + nrecv_sm]) {
              if (loop > 0 && loop % FLUSH_INTERVAL == 0) recv_proxy_channels[i].flush();
              recv_proxy_channels[i].signal();
            }
          }
          __syncthreads();
        } while (min_ready <= loop);
      }

      if (nsend_sm + nsend_proxy > 0) {
        if (node_type == 0) { // root
          for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].signal();
          for (int i = tid; i < nsend_proxy; i += blockDim.x) {
            uint64_t d_start = data_start + loop * nelem_per_send;
            uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
            if (loop == 0) send_proxy_channels[i].wait();
            else if (loop % FLUSH_INTERVAL == 0) send_proxy_channels[i].flush();
            send_proxy_channels[i].putWithSignal(d_start * sizeof(int), size * sizeof(int));
          }
        } else {
          // assert nsend_sm + nsend_proxy == 1
          if (tid == 0) {
            if (nsend_sm == 1) {
              send_sm_channels[0].signal();
            } else {
              if (pending_sends == max_pending_sends) {
                send_proxy_channels[0].wait();
                pending_sends -= 1 + send_proxy_channels[0].poll(pending_sends - 1);
              }

              uint64_t s_start = (loop % max_pending_sends) * nelem_per_send;
              uint64_t d_start = data_start + loop * nelem_per_send;
              uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
              if (loop > 0 && loop % FLUSH_INTERVAL == 0) send_proxy_channels[0].flush();
              send_proxy_channels[0].putWithSignal(s_start * sizeof(int), d_start * sizeof(int), size * sizeof(int));
              ++pending_sends;
            }
          }
          __syncthreads();
        }
      }
    }
    for (int i = tid; i < nrecv_sm; i += blockDim.x) recv_sm_channels[i].signal();
    if (node_type == 0) { // root
      for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].wait();
    } else {
      if (tid == 0) {
        if (nsend_sm == 1) send_sm_channels[0].wait();
        else if (nsend_proxy == 1) {
          do {
            pending_sends -= send_proxy_channels[0].poll(pending_sends);
          } while (pending_sends > 0);
        }
      }
    }
  } else {
    // assert nrecv_sm + nrecv_proxy <= 1
    if (nrecv_sm == 0 && nrecv_proxy == 0) {
      for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].signal(nloops);
      for (int sloop = 0; sloop < nloops; ++sloop) {
        uint64_t d_start = data_start + sloop * nelem_per_send;
        uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
        for (int i = tid; i < nsend_proxy; i += blockDim.x) {
          if (sloop == 0) send_proxy_channels[i].wait();
          send_proxy_channels[i].putWithSignalAndFlush(d_start * sizeof(int), size * sizeof(int));
        }
      }
    } else {
      if (tid == 0 && nrecv_proxy == 1) recv_proxy_channels[0].signal();
      int sloop = 0;
      __shared__ int ready;
      if (tid == 0) ready = 0;
      while (sloop < nloops) {
        if (tid == 0) {
          if (nrecv_sm == 1) {
            recv_sm_channels[0].wait();
            ready = sloop + 1 + recv_sm_channels[0].poll(nloops - sloop - 1);
          } else {
            recv_proxy_channels[0].wait();
            ready = sloop + 1 + recv_proxy_channels[0].poll(nloops - sloop - 1);
          }
        }
        __syncthreads();
        const int ready_loop = ready;
        do {
          uint64_t d_start = data_start + sloop * nelem_per_send;
          uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
          if (nrecv_sm == 1) recv_sm_channels[0].get(d_start * sizeof(int), size * sizeof(int), tid, blockDim.x);
          __syncthreads();
          for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].signal();
          for (int i = tid; i < nsend_proxy; i += blockDim.x) {
            if (sloop == 0) send_proxy_channels[i].wait();
            else if (sloop % FLUSH_INTERVAL == 0) send_proxy_channels[i].flush();
            send_proxy_channels[i].putWithSignal(d_start * sizeof(int), size * sizeof(int));
          }
          ++sloop;
        } while (sloop < ready_loop);
      }
    }
    if (tid == 0 && nrecv_sm == 1) recv_sm_channels[0].signal(); // `signal` to ensure sender wait until `get` finishes
    for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].wait();
  }
  for (int i = tid; i < nrecv_proxy; i += blockDim.x) recv_proxy_channels[i].flush();
  for (int i = tid; i < nsend_proxy; i += blockDim.x) send_proxy_channels[i].flush();
}

/// Call threadblockCall.
/// SM channel scratches: recv_scratches[block_scratch_starts[bid], ..., block_scratch_starts[bid] + nrecvs_sm[bid] - 1]
/// Proxy channel scratches: recv_scratches[block_scratch_starts[bid] + nrecvs_sm[bid], ..., block_scratch_starts[bid] + nrecvs_sm[bid] + nrecvs_proxy[bid] - 1]
extern "C" __global__ void __launch_bounds__(1024)
    KERNEL(mscclpp::SmChannelDeviceHandle* recv_sm_channels, mscclpp::SmChannelDeviceHandle* send_sm_channels,
           mscclpp::SimpleProxyChannelDeviceHandle* recv_proxy_channels, mscclpp::SimpleProxyChannelDeviceHandle* send_proxy_channels,
           int** recv_scratches, int* block_recv_sm_ch_starts, int* block_send_sm_ch_starts,
           int* block_recv_proxy_ch_starts, int* block_send_proxy_ch_starts,
           int* block_scratch_starts,
           int* nrecvs_sm, int* nsends_sm, int* nrecvs_proxy, int* nsends_proxy,
           char* node_types, const uint64_t scratch_size, int* data, 
           const uint64_t* data_start, const uint64_t nelem_per_send, const uint64_t* nelem_total) {
  const int bid = blockIdx.x;

  threadblockCall(recv_sm_channels == nullptr ? nullptr : &recv_sm_channels[block_recv_sm_ch_starts[bid]], 
                  send_sm_channels == nullptr ? nullptr : &send_sm_channels[block_send_sm_ch_starts[bid]],
                  recv_proxy_channels == nullptr ? nullptr : &recv_proxy_channels[block_recv_proxy_ch_starts[bid]], 
                  send_proxy_channels == nullptr ? nullptr : &send_proxy_channels[block_send_proxy_ch_starts[bid]],
                  &recv_scratches[block_scratch_starts[bid]], nrecvs_sm[bid], nsends_sm[bid], nrecvs_proxy[bid], nsends_proxy[bid],
                  node_types[bid], scratch_size, data,
                  data_start[bid], nelem_per_send, nelem_total[bid]);
}