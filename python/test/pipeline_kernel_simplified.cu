#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <mscclpp/sm_channel_device.hpp>
#include <mscclpp/proxy_channel_device.hpp>

// BEGIN_DEFINES //

#ifndef PARAMETRIZE
#define KERNEL pipeline_schedule
#define N_PEERS 8
#endif

# define FLUSH_INTERVAL 1000

// END_DEFINES //

/// The call is a single node in the tree.
///
/// Syncronization: It is guaranteed that in allreduce and allgather, after the kernel finishes,
/// the memory is safe to be written. However, the kernel immediately starts to write remote
/// memory once launched. There are two cases:
/// Allreduce/ReduceScatter - The kernel writes to remote scratch buffer in the beginning.
/// As long as the remote peer has finished initializing scratch buffer, the write is safe.
/// Allgather - The kernel (proxy) waits for remote signal before writing to remote.
///
/// @param recv_sm_channels SM channels for recv.
/// @param send_sm_channels SM channels for send.
/// @param recv_proxy_channels Proxy channels for recv.
/// @param send_proxy_channels Proxy channels for send.
/// @param recv_scratches Scratch buffers for each recv_sm_channels, len(recv_scratches) == len(recv_sm_channels) + len(recv_proxy_channels)
/// @param nrecv_sm Num of valid recv_sm_channels (nrecv_sm + nrecv_proxy <= 1 if broadcast).
/// @param nsend_sm Num of valid send_sm_channels (nsend_sm + nsend_proxy <= 1 if reduce).
/// @param nrecv_proxy Num of valid recv_proxy_channels (nrecv_sm + nrecv_proxy <= 1 if broadcast).
/// @param nsend_proxy Num of valid send_proxy_channels (nsend_sm + nsend_proxy <= 1 if reduce).
/// @param node_type <0: reduce node; =0: root node; >0: broadcast node.
/// The send channels of broadcast and root nodes write to data buffer.
/// The send channels of reduce node write to scratch buffer.
/// @param scratch_size Max num of elements in scratch buffer for each recv channel (ignore if not reduce).
/// scratch_size must be greater than nelem_per_send
/// @param data Data buffer.
/// @param data_start The data buffer start.
/// @param nelem_per_send Num of elements in each send.
/// @param nelem_total Total num of elements need to be send/recv.
MSCCLPP_DEVICE_INLINE void
    threadblockCall(mscclpp::SmChannelDeviceHandle* recv_sm_channels, mscclpp::SmChannelDeviceHandle* send_sm_channels,
                    mscclpp::SimpleProxyChannelDeviceHandle* recv_proxy_channels, mscclpp::SimpleProxyChannelDeviceHandle* send_proxy_channels,
                    int** recv_scratches, const int nrecv_sm, const int nsend_sm, const int nrecv_proxy, const int nsend_proxy,
                    const char node_type, const uint64_t scratch_size, int* data,
                    const uint64_t data_start, const uint64_t nelem_per_send, const uint64_t nelem_total) {
  const int tid = threadIdx.x;

  const int nloops = (nelem_total + nelem_per_send - 1) / nelem_per_send; // ceiling division

  if (node_type <= 0) {
    int reduced[N_PEERS] = {};
    __shared__ int ready[N_PEERS];
    __shared__ int pending_sends; // Only reduce node needs to track pending sends.
                                  // Reduce node has at most one send peer.

    #pragma unroll
    for (int i = tid; i < N_PEERS; i += blockDim.x) ready[i] = 0;
    if (tid == 0) pending_sends = 0;
    __syncthreads();

    const int max_pending_sends = scratch_size / nelem_per_send;

    int min_ready = 0;
    for (int loop = 0; loop < nloops; ++loop) {
      if (nrecv_sm + nrecv_proxy > 0) {
        const uint64_t s_start = (loop % max_pending_sends) * nelem_per_send;
        const uint64_t d_start = data_start + loop * nelem_per_send;
        const uint64_t nElem = min(nelem_per_send, data_start + nelem_total - d_start);
        const uint64_t nElem4 = nElem / 4;
        const uint64_t nLastElem = nElem % 4;
        int4* const data4 = (int4*) &data[d_start];

        do {
          if (min_ready <= loop) {
            bool hasUpdate;
            do {
              hasUpdate = false;
              for (int i = tid; i < nrecv_sm + nrecv_proxy; i += blockDim.x) {
                const int ready_loop = ready[i];
                if (ready_loop < nloops) {
                  int update = (i < nrecv_sm ? recv_sm_channels[i].poll(nloops - ready_loop) :
                                               recv_proxy_channels[i - nrecv_sm].poll(nloops - ready_loop));
                  if (update > 0) hasUpdate = true;
                  ready[i] = ready_loop + update;
                }
              }
            } while (!__syncthreads_or(hasUpdate));
          }

          min_ready = nloops;
          bool chHasUpdate[N_PEERS];
          for (int i = 0; i < nrecv_sm + nrecv_proxy; ++i) {
            const int ready_loop = ready[i];
            if (ready_loop < min_ready) min_ready = ready_loop;
            if (ready_loop > loop && reduced[i] == loop) {
              chHasUpdate[i] = true;
              int4* scratch4 = (int4*) &recv_scratches[i][s_start];
              for (uint64_t offset = tid; offset < nElem4; offset += blockDim.x) {
                data4[offset].x += scratch4[offset].x;
                data4[offset].y += scratch4[offset].y;
                data4[offset].z += scratch4[offset].z;
                data4[offset].w += scratch4[offset].w;
              }
              for (uint64_t offset = tid; offset < nLastElem; offset += blockDim.x) {
                data[d_start + nElem4 * 4 + offset] += recv_scratches[i][s_start + nElem4 * 4 + offset];
              }
              ++reduced[i];
            } else {
              chHasUpdate[i] = false;
            }
          }
          __syncthreads();
          for (int i = tid; i < nrecv_sm + nrecv_proxy; i += blockDim.x) {
            if (chHasUpdate[i]) {
              if (i < nrecv_sm) recv_sm_channels[i].signal();
              else {
                if (loop > 0 && loop % FLUSH_INTERVAL == 0) recv_proxy_channels[i - nrecv_sm].flush();
                recv_proxy_channels[i - nrecv_sm].signal();
              }
            }
          }
          __syncthreads();
        } while (min_ready <= loop);
      }

      if (nsend_sm + nsend_proxy > 0) {
        if (node_type == 0) { // root
          for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].signal();
          for (int i = tid; i < nsend_proxy; i += blockDim.x) {
            uint64_t d_start = data_start + loop * nelem_per_send;
            uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
            if (loop == 0) send_proxy_channels[i].wait();
            else if (loop % FLUSH_INTERVAL == 0) send_proxy_channels[i].flush();
            send_proxy_channels[i].putWithSignal(d_start * sizeof(int), size * sizeof(int));
          }
        } else {
          // assert nsend_sm + nsend_proxy == 1
          int psends = pending_sends;
          if (psends == max_pending_sends) {
            if (tid == 0) {
              if (nsend_sm == 1) {
                send_sm_channels[0].wait();
                psends -= 1 + send_sm_channels[0].poll(psends - 1);
              } else {
                send_proxy_channels[0].wait();
                psends -= 1 + send_proxy_channels[0].poll(psends - 1);
              }
            }
            __syncthreads();
          }

          uint64_t s_start = (loop % max_pending_sends) * nelem_per_send;
          uint64_t d_start = data_start + loop * nelem_per_send;
          uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
          if (nsend_sm == 1) {
            send_sm_channels[0].put(s_start * sizeof(int), d_start * sizeof(int), size * sizeof(int), tid, blockDim.x);
            __syncthreads();
            if (tid == 0) {
              send_sm_channels[0].signal();
              pending_sends = psends + 1;
            }
          } else if (tid == 0) {
            if (loop > 0 && loop % FLUSH_INTERVAL == 0) send_proxy_channels[0].flush();
            send_proxy_channels[0].putWithSignal(s_start * sizeof(int), d_start * sizeof(int), size * sizeof(int));
            pending_sends = psends + 1;
          }
          __syncthreads();
        }
      }
    }
    if (node_type == 0) { // root
      for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].wait();
    } else {
      if (tid == 0 && nsend_sm + nsend_proxy == 1) {
        int psends = pending_sends;
        do {
          psends -= (nsend_sm == 1 ? send_sm_channels[0].poll(psends) : 
                                     send_proxy_channels[0].poll(psends));
        } while (psends > 0);
      }
    }
  } else {
    // assert nrecv_sm + nrecv_proxy <= 1
    if (nrecv_sm == 0 && nrecv_proxy == 0) {
      for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].signal(nloops);
      for (int sloop = 0; sloop < nloops; ++sloop) {
        uint64_t d_start = data_start + sloop * nelem_per_send;
        uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
        for (int i = tid; i < nsend_proxy; i += blockDim.x) {
          if (sloop == 0) send_proxy_channels[i].wait();
          else if (sloop % FLUSH_INTERVAL == 0) send_proxy_channels[i].flush();
          send_proxy_channels[i].putWithSignal(d_start * sizeof(int), size * sizeof(int));
        }
      }
    } else {
      if (tid == 0 && nrecv_proxy == 1) recv_proxy_channels[0].signal();
      int sloop = 0;
      __shared__ int ready;
      if (tid == 0) ready = 0;
      while (sloop < nloops) {
        if (tid == 0) {
          if (nrecv_sm == 1) {
            recv_sm_channels[0].wait();
            ready = sloop + 1 + recv_sm_channels[0].poll(nloops - sloop - 1);
          } else {
            recv_proxy_channels[0].wait();
            ready = sloop + 1 + recv_proxy_channels[0].poll(nloops - sloop - 1);
          }
        }
        __syncthreads();
        const int ready_loop = ready;
        do {
          uint64_t d_start = data_start + sloop * nelem_per_send;
          uint64_t size = min(nelem_per_send, data_start + nelem_total - d_start);
          if (nrecv_sm == 1) recv_sm_channels[0].get(d_start * sizeof(int), size * sizeof(int), tid, blockDim.x);
          __syncthreads();
          for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].signal();
          for (int i = tid; i < nsend_proxy; i += blockDim.x) {
            if (sloop == 0) send_proxy_channels[i].wait();
            else if (sloop % FLUSH_INTERVAL == 0) send_proxy_channels[i].flush();
            send_proxy_channels[i].putWithSignal(d_start * sizeof(int), size * sizeof(int));
          }
          ++sloop;
        } while (sloop < ready_loop);
      }
    }
    if (tid == 0 && nrecv_sm == 1) recv_sm_channels[0].signal(); // `signal` to ensure sender wait until `get` finishes
    for (int i = tid; i < nsend_sm; i += blockDim.x) send_sm_channels[i].wait();
  }
  for (int i = tid; i < nrecv_proxy; i += blockDim.x) recv_proxy_channels[i].flush();
  for (int i = tid; i < nsend_proxy; i += blockDim.x) send_proxy_channels[i].flush();
}

/// Call threadblockCall.
/// SM channel scratches: recv_scratches[block_scratch_starts[bid], ..., block_scratch_starts[bid] + nrecvs_sm[bid] - 1]
/// Proxy channel scratches: recv_scratches[block_scratch_starts[bid] + nrecvs_sm[bid], ..., block_scratch_starts[bid] + nrecvs_sm[bid] + nrecvs_proxy[bid] - 1]
extern "C" __global__ void __launch_bounds__(1024)
    KERNEL(mscclpp::SmChannelDeviceHandle* recv_sm_channels, mscclpp::SmChannelDeviceHandle* send_sm_channels,
           mscclpp::SimpleProxyChannelDeviceHandle* recv_proxy_channels, mscclpp::SimpleProxyChannelDeviceHandle* send_proxy_channels,
           int** recv_scratches, int* block_recv_sm_ch_starts, int* block_send_sm_ch_starts,
           int* block_recv_proxy_ch_starts, int* block_send_proxy_ch_starts,
           int* block_scratch_starts,
           int* nrecvs_sm, int* nsends_sm, int* nrecvs_proxy, int* nsends_proxy,
           char* node_types, const uint64_t scratch_size, int* data, 
           const uint64_t* data_start, const uint64_t nelem_per_send, const uint64_t* nelem_total) {
  const int bid = blockIdx.x;

  threadblockCall(recv_sm_channels == nullptr ? nullptr : &recv_sm_channels[block_recv_sm_ch_starts[bid]], 
                  send_sm_channels == nullptr ? nullptr : &send_sm_channels[block_send_sm_ch_starts[bid]],
                  recv_proxy_channels == nullptr ? nullptr : &recv_proxy_channels[block_recv_proxy_ch_starts[bid]], 
                  send_proxy_channels == nullptr ? nullptr : &send_proxy_channels[block_send_proxy_ch_starts[bid]],
                  &recv_scratches[block_scratch_starts[bid]], nrecvs_sm[bid], nsends_sm[bid], nrecvs_proxy[bid], nsends_proxy[bid],
                  node_types[bid], scratch_size, data,
                  data_start[bid], nelem_per_send, nelem_total[bid]);
}